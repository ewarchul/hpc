
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define THREADS_PER_BLOCK 768
#define ARRAY_SIZE THREADS_PER_BLOCK * 1024
#define OPTIM 0

static void HandleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__ ))

__global__ void reverseArray(int *inArray, int *outArray)
{
    int inOffset = blockDim.x * blockIdx.x;

//		int inIndex = inOffset + blockIdx.x;
//                             ^
//                             print'ujac inIndex zauwazylem w gdb, ze
//                             w obrebie bloku z threadu na thread nie
//                             zmienienia sie jego wartosc, a co wiecej:
//                             wychodzimy poza adres pamieci (cuda-memcheck)
//                             np. dla bloku 1023.
//														

		int inIndex = inOffset + threadIdx.x; 
//                            ^ poprawne przesuniecie

//    int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
//		int outIndex = outOffset + (blockDim.x - 1 - blockIdx.x);
//                                                  ^ analogicznie jak wyzej.
		int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
		int outIndex = outOffset + (blockDim.x - 1 - threadIdx.x);

    outArray[outIndex] = inArray[inIndex];
}

__global__ void reverseArrayOptim(int *inArray, int *outArray) {
		__shared__ int shared_memory[THREADS_PER_BLOCK];
    int inOffset = blockDim.x * blockIdx.x;
		int inIndex = inOffset + threadIdx.x; 
		shared_memory[blockDim.x - 1 - threadIdx.x] = inArray[inIndex];
//                                   ^ reverse
		__syncthreads();
		int outOffset = blockDim.x * (gridDim.x - 1 - blockIdx.x);
		int outIndex = outOffset + threadIdx.x; 
    outArray[outIndex] = shared_memory[threadIdx.x];
}

void print_vec(int n, int *array) {
	printf(" array =  { ");
	for (int i = 0; i < n; ++i) {
		printf("%d, ", array[i]);
	}
	printf("}\n");
}

int main(void)
{
    int *hostArray;
    int *devInArray, *devOutArray;
    int numBlocks = ARRAY_SIZE / THREADS_PER_BLOCK;
    size_t memSize = ARRAY_SIZE * sizeof(int);
    hostArray = (int *)malloc(memSize);
		for (int i = 0; i < ARRAY_SIZE; i++) {
        hostArray[i] = i;
    }

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);

    HANDLE_ERROR(hipMalloc((void **)&devInArray, memSize));
    HANDLE_ERROR(hipMalloc((void **)&devOutArray, memSize));
    HANDLE_ERROR(hipMemcpy(devInArray, hostArray, memSize, hipMemcpyHostToDevice));
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(THREADS_PER_BLOCK);

		#if OPTIM
    	reverseArrayOptim<<<dimGrid, dimBlock>>>  (devInArray, devOutArray);
		#else 
    	reverseArray<<<dimGrid, dimBlock>>>  (devInArray, devOutArray);
		#endif

    HANDLE_ERROR(hipMemcpy(hostArray, devOutArray, memSize, hipMemcpyDeviceToHost));

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float exec_time = 0;

		hipEventElapsedTime(&exec_time, start, stop);
		#if OPTIM
			printf("[GPU/OPTIM] Execution time (ms): %3.1f\n", exec_time);
		#else 
			printf("[GPU] Execution time (ms): %3.1f\n", exec_time);
		#endif

    for (int i = 0; i < ARRAY_SIZE; i++) {
        assert(hostArray[i] == ARRAY_SIZE - 1 - i);
    }

    HANDLE_ERROR(hipFree(devInArray));
    HANDLE_ERROR(hipFree(devOutArray));

    free(hostArray);

    printf("Correct!\n");
    return 0;
}

